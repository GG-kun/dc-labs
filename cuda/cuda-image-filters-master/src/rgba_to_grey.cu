#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

#include <iostream>
#include <string>

// Function calling the kernel to operate
void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, 
                  size_t numRows, size_t numCols);

//include the definitions of the above functions for this homework
#include "preprocess.cpp"

using namespace std;
using namespace cv;

// Max Threads per block in GeForce 210
#define TxB 512

__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  // The mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  // Since it does not matter the relative position of a pixel
  // the block - grid assign strategy will simply be to cover
  // all pixels secuencially in 'x' axis
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

int main(int argc, char **argv) {
  uchar4        *h_rgbaImage, *d_rgbaImage;
  unsigned char *h_greyImage, *d_greyImage;

  string input_file;
  string output_file;

  //make sure the context initializes ok
  checkCudaErrors(hipFree(0));

  switch (argc)
  {
	case 2:
	  input_file = string(argv[1]);
	  output_file = string(argv[1]);
	  break;
  case 3:
    input_file = string(argv[1]);
	  output_file = string(argv[2]);
    break;
	default:
      cerr << "Usage: ./to_bw input_file [output_filename]" << endl;
      exit(1);
  }
  //load the image and give us our input and output pointers
  preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

  //call the cuda code
  rgba_to_grey(d_rgbaImage, d_greyImage, numRows(), numCols());

  size_t numPixels = numRows()*numCols();
  checkCudaErrors(hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

  /* Output the grey image */
  Mat output(numRows(), numCols(), CV_8UC1, (void*)h_greyImage);
  //output the image
  imwrite(output_file.c_str(), output);

  /* Cleanup */
  hipFree(d_rgbaImage__);
  hipFree(d_greyImage__);

  return 0;
}
